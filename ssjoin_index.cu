#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <similarity.cuh>

__global__ void count_tokens(
    const uint32_t *records_d,
    const int cardinality,
    uint32_t *count_d,
    const float threshold)
{
    const int stride = STRIDE();
    uint32_t token_max{0}, token_count{0};
    count_d += 2; // reserve 2 cells for [token_max, token_count]

    for (int idx = IDX(); idx < cardinality; idx += stride)
    {
        auto start{records_d[idx]};
        auto size{records_d[idx + 1] - start};
        size = size + 1 - OVERLAP(threshold, size, size);
        token_count += size;

        const auto end{start + size};
        do
        {
            auto token{records_d[start]};
            token_max = token > token_max ? token : token_max;
            atomicAdd(count_d + token, 1);
        } while (++start < end);
    }
    atomicMax(count_d - 2, token_max);
    atomicAdd(count_d - 1, token_count);
}
