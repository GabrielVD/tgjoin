#include "hip/hip_runtime.h"
#include <ssjoin_verify.cuh>
#include <hip/hip_runtime_api.h>

#define FILL ((overlap_t)-1)

__global__ void verify(
    int* __restrict__ buffer_d,
    overlap_pack* __restrict__ overlap_pack_d,
    const size_t pack_count)
{
    // count of elements in buffer
    extern __shared__ int shared[];
    size_t *buffer = ((size_t*)shared) + 1;

    if (threadIdx.x == 0) { shared[0] = 0; }

    const size_t stride = STRIDE();
    int candidates = 0;
    for (size_t idx = IDX(); idx < pack_count; idx += stride)
    {
        __syncthreads();
        size_t overlap_index = idx * OVERLAP_PACK_SIZE;
        auto pack = atomicExch(overlap_pack_d + idx, 0);

        while (1)
        {
            while (pack != 0)
            {
                overlap_t overlap = pack & FILL;
                if (overlap != 0 && overlap != FILL)
                {
                    int old_count = atomicAdd(shared, 1);
                    if (old_count >= blockDim.x) { break; }
                    buffer[old_count] = overlap_index;
                }
                pack = pack >> (sizeof(overlap_t) * CHAR_BIT);
                ++overlap_index;
            }

            __syncthreads();
            if (shared[0] < blockDim.x) { break; }
            __syncthreads();
            if (threadIdx.x == 0)
            {
                candidates += blockDim.x;
                shared[0] = 0;
            }
        }
    }

    if (threadIdx.x == 0) { atomicAdd(buffer_d, shared[0] + candidates); }
}
