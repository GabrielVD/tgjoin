#include "hip/hip_runtime.h"
#include <ssjoin_verify.cuh>
#include <hip/hip_runtime_api.h>

#define FILL ((overlap_t)-1)

__global__ void verify(
    overlap_pack *overlap_pack_d,
    const int pack_count)
{
    extern __shared__ int shared[];

    if (threadIdx.x == 0) { shared[0] = 0; }

    volatile overlap_t overlap;
    const int stride = STRIDE();
    for (int idx = IDX(); idx < pack_count; idx += stride)
    {
        __syncthreads();
        auto pack = atomicExch(overlap_pack_d + idx, 0);

        while (1)
        {
            while (pack != 0)
            {
                overlap = pack & FILL;
                if (overlap == 0 || overlap != FILL)
                {
                    if (atomicAdd(shared, 1) >= blockDim.x) { break; }
                }
                pack = pack >> (sizeof(overlap_t) * CHAR_BIT);
            }

            __syncthreads();
            if (shared[0] < blockDim.x) { break; }
            if (threadIdx.x == 0) { shared[0] = 0; }
        }
        
    }
}
