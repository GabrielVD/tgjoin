#include "hip/hip_runtime.h"
#include <ssjoin.h>
#include <ssjoin_staging.h>
#include <ssjoin_index.cuh>
#include <ssjoin_filtering.cuh>
#include <helper_mem.cuh>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <similarity.cuh>

// struct kernel_config
// {
//     launch_params count_tokens;
//     launch_params make_index;
//     launch_params filter;
// };

struct pointers_t
{
    byte_t *pool_d, *pool_limit_d;
    record_t *buffer, *dataset_d, *record_map_d, *buffer_d;
};

struct streams_t
{
    hipStream_t a, b;
};

struct joinstate_t
{
    pointers_t ptr;
    ssjoin_stats stats;
    streams_t stream;
};

// struct pointers
// {
//     record_t *buffer, *buffer_d, *records_d, *token_map_d;
//     index_record *inverted_index_d;
//     uint8_t *overlap_matrix_d;
//     size_t buffer_size;
// };

// static kernel_config get_config()
// {
//     kernel_config config;

//     checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
//         &config.count_tokens.grid,
//         &config.count_tokens.block,
//         count_tokens));

//     checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
//         &config.make_index.grid,
//         &config.make_index.block,
//         make_index));

//     checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
//         &config.filter.grid,
//         &config.filter.block,
//         filter));

//     return config;
// }

static int file_read(ssjoin_stats &stats, input_info &info, record_t **buffer)
{
    if (check_filesize(info.pathname, &info.datacount) != 0
    || load_file(info.pathname, buffer, info.datacount) != 0)
    {
        stats.status = ssjoin_status::IO_ERR;
        return 1;
    }

    if (verify_dataset(*buffer, info) != 0)
    {
        stats.status = ssjoin_status::FORMAT_ERR;
        checkCudaErrors(hipHostFree(*buffer));
        *buffer = NULL;
        return 1;
    }

    return 0;
}

static size_t pool_size(const input_info &info)
{
    size_t size;
    checkCudaErrors(hipMemGetInfo(&size, NULL));
    size -= info.mem_min;

    const size_t overlap_size{BYTES_O(tri_rowstart(info.cardinality + 1))};
    if (overlap_size < size)
    {
        size = std::min(
            size,
            2 * BYTES_R(info.datacount)
                + BYTES_R(info.cardinality)
                + BYTES_I(info.cardinality * (size_t)info.avg_set_size)
                + overlap_size);
    }

    return size;
}

static void map_records_async(
    record_t *record_map,
    size_t size,
    const joinstate_t &state,
    const input_info &info,
    hipStream_t stream)
{
    record_t record_index{1};
    for (int i = 0; i < info.cardinality; i++)
    {
        record_map[i] = record_index;
        record_index += state.ptr.buffer[record_index] + 2;
    }

    checkCudaErrors(hipMemcpyAsync(
        state.ptr.record_map_d,
        record_map,
        size,
        hipMemcpyHostToDevice,
        stream));
}

static void host_to_device(joinstate_t &state, input_info &info)
{
    state.stats.pool_size = pool_size(info);
    checkCudaErrors(hipMalloc(&state.ptr.pool_d, state.stats.pool_size));

    size_t input_bytes = BYTES_R(info.datacount);
    checkCudaErrors(hipMemcpyAsync(
        state.ptr.pool_d,
        state.ptr.buffer,
        input_bytes,
        hipMemcpyHostToDevice,
        state.stream.a));

    state.ptr.record_map_d = (record_t*)(state.ptr.pool_d + input_bytes);
    state.ptr.pool_limit_d = state.ptr.pool_d + state.stats.pool_size;
    state.ptr.buffer_d = state.ptr.record_map_d + info.cardinality;

    const size_t cardinality_bytes = BYTES_R(info.cardinality);
    checkCudaErrors(hipMemsetAsync(
        state.ptr.buffer_d,
        0,
        input_bytes - 2 * cardinality_bytes,
        state.stream.b));

    record_t *record_map = (record_t *)malloc(cardinality_bytes);
    map_records_async(record_map, cardinality_bytes, state, info, state.stream.a);
    
    info.overlap_factor = OVERLAP_FAC(info.threshold);

    state.ptr.dataset_d = (record_t*)state.ptr.pool_d;
    checkCudaErrors(hipDeviceSynchronize());
    free(record_map);
}

// static void indexing(
//     ssjoin_stats &stats,
//     const kernel_config &config,
//     record_t cardinality,
//     pointers &p,
//     const float overlap_factor)
// {
//     count_tokens<<<config.count_tokens.grid, config.count_tokens.block>>>(
//         p.records_d,
//         cardinality,
//         p.buffer_d,
//         overlap_factor);

//     // copy [token_max, token_count]
//     checkCudaErrors(
//         hipMemcpyAsync(p.buffer, p.buffer_d, BYTES_R(2), hipMemcpyDeviceToHost));
//     checkCudaErrors(hipDeviceSynchronize());

//     // limit of starting indexes
//     stats.token_map_limit = p.buffer[0] + 1;

//     checkCudaErrors(hipMalloc(&p.token_map_d, BYTES_R(stats.token_map_limit + 1)));
//     prefix_sum(p.buffer_d + 2, stats.token_map_limit + 1, p.token_map_d);

//     stats.indexed_entries = p.buffer[1];
//     checkCudaErrors(hipMalloc(&p.inverted_index_d, BYTES_INDEX(stats.indexed_entries)));

//     make_index<<<config.make_index.grid, config.make_index.block>>>(
//         p.records_d,
//         cardinality,
//         p.token_map_d,
//         overlap_factor,
//         p.buffer_d + 3, // starting address of token count
//         p.inverted_index_d);

//     checkCudaErrors(hipDeviceSynchronize());
// }

// static void filtering(
//     ssjoin_stats &stats,
//     const kernel_config &config,
//     const input_info &info,
//     pointers &p,
//     const float overlap_factor)
// {
//     checkCudaErrors(hipMemGetInfo(&stats.matrix_bytesize, NULL));
//     record_t id_limit = tri_maxfit((stats.matrix_bytesize - info.mem_min)
//                                 / sizeof(*p.overlap_matrix_d));
//     id_limit = std::min(id_limit, info.cardinality);
//     stats.matrix_size = tri_rowstart(id_limit);
//     stats.matrix_bytesize = stats.matrix_size * sizeof(*p.overlap_matrix_d);
//     checkCudaErrors(hipMalloc(&p.overlap_matrix_d, stats.matrix_bytesize));

//     record_t id_start{1};
//     auto dirty_bytes{stats.matrix_bytesize};
//     auto *matrix_tip_d{p.overlap_matrix_d - tri_rowstart(id_start)};
//     do
//     {
//         checkCudaErrors(hipMemsetAsync(p.overlap_matrix_d, 0, dirty_bytes));
//         filter<<<config.filter.grid, config.filter.block>>>(
//             p.records_d,
//             id_start,
//             id_limit,
//             p.token_map_d,
//             stats.token_map_limit,
//             p.inverted_index_d,
//             info.threshold,
//             overlap_factor,
//             matrix_tip_d,
//             p.buffer_d);

//         ++stats.iterations;
//         id_start = id_limit;
//         id_limit = tri_maxfit(stats.matrix_size + tri_rowstart(id_start));
//         id_limit = std::min(id_limit, info.cardinality);
//         matrix_tip_d = p.overlap_matrix_d - tri_rowstart(id_start);
//         dirty_bytes = ((matrix_tip_d + tri_rowstart(id_limit)) - p.overlap_matrix_d)
//             * sizeof(*p.overlap_matrix_d);
//         checkCudaErrors(hipDeviceSynchronize());
//     } while (id_start < info.cardinality);
// }

ssjoin_stats run_join(input_info info)
{
    joinstate_t state;

    {
        auto start{NOW()};
        if (file_read(state.stats, info, &state.ptr.buffer) != 0)
        {
            return state.stats;
        }
        state.stats.read_ms = TIME_MS(NOW() - start);
        info.print(stderr);
    }

    {
        auto start{NOW()};
        checkCudaErrors(hipStreamCreate(&state.stream.a));
        checkCudaErrors(hipStreamCreate(&state.stream.b));
        host_to_device(state, info);
        state.stats.host2device_ms = TIME_MS(NOW() - start);
    }

    // kernel_config config{get_config()};
    // pointers p;
    // float overlap_factor;

    // {
    //     auto start{NOW()};
    //     host_to_device(input, info, p, overlap_factor);
    //     stats.host2device_ms = TIME_MS(NOW() - start);
    // }

    // {
    //     auto start{NOW()};
    //     indexing(stats, config, info.cardinality, p, overlap_factor);
    //     stats.indexing_ms = TIME_MS(NOW() - start);
    // }

    // {
    //     checkCudaErrors(hipMemset(p.buffer_d, 0, BYTES_R(2)));
    //     auto start{NOW()};
    //     filtering(stats, config, info, p, overlap_factor);
    //     stats.filtering_ms = TIME_MS(NOW() - start);
    //     checkCudaErrors(hipMemcpy(p.buffer, p.buffer_d, BYTES_R(2), hipMemcpyDeviceToHost));
    //     stats.token_probes = p.buffer[0];
    //     stats.index_probes = p.buffer[1];
    // }
    
    checkCudaErrors(hipFree(state.ptr.pool_d));
    checkCudaErrors(hipHostFree(state.ptr.buffer));
    state.stats.status = ssjoin_status::SUCCESS;
    return state.stats;
}
