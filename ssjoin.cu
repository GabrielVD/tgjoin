#include "hip/hip_runtime.h"
#include <ssjoin.h>
#include <ssjoin_staging.h>
#include <ssjoin_index.cuh>
#include <ssjoin_filtering.cuh>
#include <helper_mem.cuh>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <similarity.cuh>

// struct kernel_config
// {
//     launch_params count_tokens;
//     launch_params make_index;
//     launch_params filter;
// };

struct pointers_t
{
    record_t *buffer, *pool_d;
};

struct streams_t
{
    hipStream_t a, b;
};

struct joinstate_t
{
    pointers_t ptr;
    ssjoin_stats stats;
    streams_t stream;
};

// struct pointers
// {
//     record_t *buffer, *buffer_d, *records_d, *token_map_d;
//     index_record *inverted_index_d;
//     uint8_t *overlap_matrix_d;
//     size_t buffer_size;
// };

// static kernel_config get_config()
// {
//     kernel_config config;

//     checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
//         &config.count_tokens.grid,
//         &config.count_tokens.block,
//         count_tokens));

//     checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
//         &config.make_index.grid,
//         &config.make_index.block,
//         make_index));

//     checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
//         &config.filter.grid,
//         &config.filter.block,
//         filter));

//     return config;
// }

static int file_read(ssjoin_stats &stats, input_info &info, record_t **buffer)
{
    if (check_filesize(info.pathname, &info.datacount) != 0
    || load_file(info.pathname, buffer, info.datacount) != 0)
    {
        stats.status = ssjoin_status::IO_ERR;
        return 1;
    }

    if (verify_dataset(*buffer, info) != 0)
    {
        stats.status = ssjoin_status::FORMAT_ERR;
        checkCudaErrors(hipHostFree(*buffer));
        *buffer = NULL;
        return 1;
    }

    return 0;
}

static void host_to_device(joinstate_t &state, const input_info &info)
{
    const size_t input_bytes{BYTES_R(info.datacount)};
    {
        checkCudaErrors(hipMemGetInfo(&state.stats.pool_size, NULL));
        state.stats.pool_size -= info.mem_min;

        const auto overlap_size{BYTES_O(tri_rowstart(info.cardinality + 1))};
        if (overlap_size < state.stats.pool_size)
        {
            state.stats.pool_size = std::min(
                state.stats.pool_size,
                2 * input_bytes
                    + BYTES_R(info.cardinality)
                    + BYTES_I(info.cardinality * (size_t)info.avg_set_size)
                    + overlap_size);
        }
    }

    checkCudaErrors(hipMalloc(&state.ptr.pool_d, state.stats.pool_size));
    checkCudaErrors(hipMemcpyAsync(
        state.ptr.pool_d,
        state.ptr.buffer,
        input_bytes,
        hipMemcpyHostToDevice,
        state.stream.a));
    
    // checkCudaErrors(hipMemsetAsync(p.buffer_d, 0, bytes, state.stream.b));
    // overlap_factor = OVERLAP_FAC(info.threshold);

    checkCudaErrors(hipDeviceSynchronize());
}

// static void indexing(
//     ssjoin_stats &stats,
//     const kernel_config &config,
//     record_t cardinality,
//     pointers &p,
//     const float overlap_factor)
// {
//     count_tokens<<<config.count_tokens.grid, config.count_tokens.block>>>(
//         p.records_d,
//         cardinality,
//         p.buffer_d,
//         overlap_factor);

//     // copy [token_max, token_count]
//     checkCudaErrors(
//         hipMemcpyAsync(p.buffer, p.buffer_d, BYTES_R(2), hipMemcpyDeviceToHost));
//     checkCudaErrors(hipDeviceSynchronize());

//     // limit of starting indexes
//     stats.token_map_limit = p.buffer[0] + 1;

//     checkCudaErrors(hipMalloc(&p.token_map_d, BYTES_R(stats.token_map_limit + 1)));
//     prefix_sum(p.buffer_d + 2, stats.token_map_limit + 1, p.token_map_d);

//     stats.indexed_entries = p.buffer[1];
//     checkCudaErrors(hipMalloc(&p.inverted_index_d, BYTES_INDEX(stats.indexed_entries)));

//     make_index<<<config.make_index.grid, config.make_index.block>>>(
//         p.records_d,
//         cardinality,
//         p.token_map_d,
//         overlap_factor,
//         p.buffer_d + 3, // starting address of token count
//         p.inverted_index_d);

//     checkCudaErrors(hipDeviceSynchronize());
// }

// static void filtering(
//     ssjoin_stats &stats,
//     const kernel_config &config,
//     const input_info &info,
//     pointers &p,
//     const float overlap_factor)
// {
//     checkCudaErrors(hipMemGetInfo(&stats.matrix_bytesize, NULL));
//     record_t id_limit = tri_maxfit((stats.matrix_bytesize - info.mem_min)
//                                 / sizeof(*p.overlap_matrix_d));
//     id_limit = std::min(id_limit, info.cardinality);
//     stats.matrix_size = tri_rowstart(id_limit);
//     stats.matrix_bytesize = stats.matrix_size * sizeof(*p.overlap_matrix_d);
//     checkCudaErrors(hipMalloc(&p.overlap_matrix_d, stats.matrix_bytesize));

//     record_t id_start{1};
//     auto dirty_bytes{stats.matrix_bytesize};
//     auto *matrix_tip_d{p.overlap_matrix_d - tri_rowstart(id_start)};
//     do
//     {
//         checkCudaErrors(hipMemsetAsync(p.overlap_matrix_d, 0, dirty_bytes));
//         filter<<<config.filter.grid, config.filter.block>>>(
//             p.records_d,
//             id_start,
//             id_limit,
//             p.token_map_d,
//             stats.token_map_limit,
//             p.inverted_index_d,
//             info.threshold,
//             overlap_factor,
//             matrix_tip_d,
//             p.buffer_d);

//         ++stats.iterations;
//         id_start = id_limit;
//         id_limit = tri_maxfit(stats.matrix_size + tri_rowstart(id_start));
//         id_limit = std::min(id_limit, info.cardinality);
//         matrix_tip_d = p.overlap_matrix_d - tri_rowstart(id_start);
//         dirty_bytes = ((matrix_tip_d + tri_rowstart(id_limit)) - p.overlap_matrix_d)
//             * sizeof(*p.overlap_matrix_d);
//         checkCudaErrors(hipDeviceSynchronize());
//     } while (id_start < info.cardinality);
// }

ssjoin_stats run_join(input_info info)
{
    joinstate_t state;

    {
        auto start{NOW()};
        if (file_read(state.stats, info, &state.ptr.buffer) != 0)
        {
            return state.stats;
        }
        state.stats.read_ms = TIME_MS(NOW() - start);
        info.print(stderr);
    }

    {
        auto start{NOW()};
        checkCudaErrors(hipStreamCreate(&state.stream.a));
        checkCudaErrors(hipStreamCreate(&state.stream.b));
        host_to_device(state, info);
        state.stats.host2device_ms = TIME_MS(NOW() - start);
    }

    // kernel_config config{get_config()};
    // pointers p;
    // float overlap_factor;

    // {
    //     auto start{NOW()};
    //     host_to_device(input, info, p, overlap_factor);
    //     stats.host2device_ms = TIME_MS(NOW() - start);
    // }

    // {
    //     auto start{NOW()};
    //     indexing(stats, config, info.cardinality, p, overlap_factor);
    //     stats.indexing_ms = TIME_MS(NOW() - start);
    // }

    // {
    //     checkCudaErrors(hipMemset(p.buffer_d, 0, BYTES_R(2)));
    //     auto start{NOW()};
    //     filtering(stats, config, info, p, overlap_factor);
    //     stats.filtering_ms = TIME_MS(NOW() - start);
    //     checkCudaErrors(hipMemcpy(p.buffer, p.buffer_d, BYTES_R(2), hipMemcpyDeviceToHost));
    //     stats.token_probes = p.buffer[0];
    //     stats.index_probes = p.buffer[1];
    // }
    
    checkCudaErrors(hipFree(state.ptr.pool_d));
    checkCudaErrors(hipHostFree(state.ptr.buffer));
    state.stats.status = ssjoin_status::SUCCESS;
    return state.stats;
}
