#include <ssjoin.h>
#include <ssjoin_shared.h>
#include <ssjoin_index.cuh>
#include <helper_mem.h>
#include <hip/hip_runtime_api.h>

struct ssjoin_config
{
    launch_config count_tokens;
    launch_config make_index;
    // launch_config sort_index;
};

struct pointers
{
    uint32_t *buffer, *buffer_d, *records_d, *token_map_d;
    index_record *inverted_index_d;
};

static ssjoin_config launch_config()
{
    ssjoin_config config;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &config.count_tokens.grid,
        &config.count_tokens.block,
        count_tokens));

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &config.make_index.grid,
        &config.make_index.block,
        make_index));

    // checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
    //     &config.sort_index.grid,
    //     &config.sort_index.block,
    //     sort_index));

    return config;
}

ssjoin_stats run_join(const uint32_t *input, size_t input_size, dataset_stats dstats)
{
    ssjoin_stats stats;
    ssjoin_config config{launch_config()};
    pointers p;
    size_t buffer_size;

    {
        auto start{NOW()};
        transfer_records_async(
            &p.buffer, &p.records_d, buffer_size,
            input, input_size, dstats.cardinality);

        const auto bytes{BYTES_U(buffer_size)};
        checkCudaErrors(hipMalloc(&p.buffer_d, bytes));
        checkCudaErrors(hipMemsetAsync(p.buffer_d, 0, bytes));
        
        checkCudaErrors(hipDeviceSynchronize());
        stats.host2device_ms = TIME_MS(NOW() - start);
    }

    {
        auto start{NOW()};
        count_tokens<<<config.count_tokens.grid, config.count_tokens.block>>>(
            p.records_d, dstats.cardinality, p.buffer_d, 0.9f);
        
        // copy [token_max, token_count]
        checkCudaErrors(
            hipMemcpyAsync(p.buffer, p.buffer_d, BYTES_U(2), hipMemcpyDeviceToHost));
        checkCudaErrors(hipDeviceSynchronize());

        stats.token_map_size = p.buffer[0] + 2;

        checkCudaErrors(hipMalloc(&p.token_map_d, BYTES_U(stats.token_map_size)));
        prefix_sum(p.buffer_d + 2, stats.token_map_size, p.token_map_d);

        stats.indexed_entries = p.buffer[1];
        checkCudaErrors(hipMalloc(&p.inverted_index_d, BYTES_INDEX(stats.indexed_entries)));

        make_index<<<config.make_index.grid, config.make_index.block>>>(
            p.records_d,
            dstats.cardinality,
            p.token_map_d,
            0.9f,
            p.buffer_d + 3, // starting address of token count
            p.inverted_index_d);

        // sort_index<<<config.sort_index.grid, config.sort_index.block>>>(
        //     p.token_map_d,
        //     stats.token_map_size,
        //     p.inverted_index_d);
        checkCudaErrors(hipDeviceSynchronize());

        stats.indexing_ms = TIME_MS(NOW() - start);
    }

    checkCudaErrors(hipFree(p.token_map_d));
    checkCudaErrors(hipFree(p.buffer_d));
    checkCudaErrors(hipFree(p.records_d));
    stats.status = ssjoin_status::SUCCESS;
    return stats;
}
