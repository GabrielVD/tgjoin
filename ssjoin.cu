#include "hip/hip_runtime.h"
#include <ssjoin.h>
#include <ssjoin_staging.h>
#include <ssjoin_index.cuh>
#include <ssjoin_filtering.cuh>
#include <helper_mem.cuh>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <similarity.cuh>

struct kernel_config
{
    launch_params count_tokens;
    launch_params make_index;
    launch_params filter;
};

struct pointers
{
    record_t *buffer, *buffer_d, *records_d, *token_map_d;
    index_record *inverted_index_d;
    uint8_t *overlap_matrix_d;
    size_t buffer_size;
};

static kernel_config get_config()
{
    kernel_config config;

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &config.count_tokens.grid,
        &config.count_tokens.block,
        count_tokens));

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &config.make_index.grid,
        &config.make_index.block,
        make_index));

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &config.filter.grid,
        &config.filter.block,
        filter));

    return config;
}

static void host_to_device(
    const record_t *input,
    const input_info &info,
    pointers &p,
    float &overlap_factor)
{
    transfer_records_async(
        &p.buffer, &p.records_d, p.buffer_size,
        input, info.data_size, info.cardinality);

    const auto bytes{BYTES_U(p.buffer_size)};
    checkCudaErrors(hipMalloc(&p.buffer_d, bytes));
    checkCudaErrors(hipMemsetAsync(p.buffer_d, 0, bytes));
    overlap_factor = OVERLAP_FAC(info.threshold);

    checkCudaErrors(hipDeviceSynchronize());
}

static void indexing(
    ssjoin_stats &stats,
    const kernel_config &config,
    int cardinality,
    pointers &p,
    const float overlap_factor)
{
    count_tokens<<<config.count_tokens.grid, config.count_tokens.block>>>(
        p.records_d,
        cardinality,
        p.buffer_d,
        overlap_factor);

    // copy [token_max, token_count]
    checkCudaErrors(
        hipMemcpyAsync(p.buffer, p.buffer_d, BYTES_U(2), hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    // limit of starting indexes
    stats.token_map_limit = p.buffer[0] + 1;

    checkCudaErrors(hipMalloc(&p.token_map_d, BYTES_U(stats.token_map_limit + 1)));
    prefix_sum(p.buffer_d + 2, stats.token_map_limit + 1, p.token_map_d);

    stats.indexed_entries = p.buffer[1];
    checkCudaErrors(hipMalloc(&p.inverted_index_d, BYTES_INDEX(stats.indexed_entries)));

    make_index<<<config.make_index.grid, config.make_index.block>>>(
        p.records_d,
        cardinality,
        p.token_map_d,
        overlap_factor,
        p.buffer_d + 3, // starting address of token count
        p.inverted_index_d);

    checkCudaErrors(hipDeviceSynchronize());
}

static void filtering(
    ssjoin_stats &stats,
    const kernel_config &config,
    const input_info &info,
    pointers &p,
    const float overlap_factor)
{
    checkCudaErrors(hipMemGetInfo(&stats.matrix_bytesize, NULL));
    int id_limit = tri_maxfit((stats.matrix_bytesize - info.mem_min)
                                / sizeof(*p.overlap_matrix_d));
    id_limit = std::min(id_limit, info.cardinality);
    stats.matrix_size = tri_rowstart(id_limit);
    stats.matrix_bytesize = stats.matrix_size * sizeof(*p.overlap_matrix_d);
    checkCudaErrors(hipMalloc(&p.overlap_matrix_d, stats.matrix_bytesize));

    int id_start{1};
    auto dirty_bytes{stats.matrix_bytesize};
    auto *matrix_tip_d{p.overlap_matrix_d - tri_rowstart(id_start)};
    do
    {
        checkCudaErrors(hipMemsetAsync(p.overlap_matrix_d, 0, dirty_bytes));
        filter<<<config.filter.grid, config.filter.block>>>(
            p.records_d,
            id_start,
            id_limit,
            p.token_map_d,
            stats.token_map_limit,
            p.inverted_index_d,
            info.threshold,
            overlap_factor,
            matrix_tip_d,
            p.buffer_d);

        ++stats.iterations;
        id_start = id_limit;
        id_limit = tri_maxfit(stats.matrix_size + tri_rowstart(id_start));
        id_limit = std::min(id_limit, info.cardinality);
        matrix_tip_d = p.overlap_matrix_d - tri_rowstart(id_start);
        dirty_bytes = ((matrix_tip_d + tri_rowstart(id_limit)) - p.overlap_matrix_d)
            * sizeof(*p.overlap_matrix_d);
        checkCudaErrors(hipDeviceSynchronize());
    } while (id_start < info.cardinality);
}

ssjoin_stats run_join(const record_t *input, input_info info)
{
    ssjoin_stats stats;
    kernel_config config{get_config()};
    pointers p;
    float overlap_factor;

    {
        auto start{NOW()};
        host_to_device(input, info, p, overlap_factor);
        stats.host2device_ms = TIME_MS(NOW() - start);
    }

    {
        auto start{NOW()};
        indexing(stats, config, info.cardinality, p, overlap_factor);
        stats.indexing_ms = TIME_MS(NOW() - start);
    }

    {
        checkCudaErrors(hipMemset(p.buffer_d, 0, BYTES_U(2)));
        auto start{NOW()};
        filtering(stats, config, info, p, overlap_factor);
        stats.filtering_ms = TIME_MS(NOW() - start);
        checkCudaErrors(hipMemcpy(p.buffer, p.buffer_d, BYTES_U(2), hipMemcpyDeviceToHost));
        stats.token_probes = p.buffer[0];
        stats.index_probes = p.buffer[1];
    }

    checkCudaErrors(hipFree(p.overlap_matrix_d));
    checkCudaErrors(hipFree(p.inverted_index_d));
    checkCudaErrors(hipFree(p.token_map_d));
    checkCudaErrors(hipFree(p.buffer_d));
    checkCudaErrors(hipFree(p.records_d));
    stats.status = ssjoin_status::SUCCESS;
    return stats;
}
